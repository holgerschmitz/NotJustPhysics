
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float dx, float *x, float *y)
{ 
  int delta = n / (blockDim.x * gridDim.x) + 1;
  int start = delta*(blockIdx.x * blockDim.x + threadIdx.x);
  int end = start + delta;
  if (end >= n) {
    end = n - 1;
  }
  for (int i = start; i < end; ++i) {
    y[i] = (x[i+1] - x[i]) / dx;
  }
}

int main(void)
{
  int N = 1<<26;
  float *x, *y;
  float dx = float(2*M_PI/N);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = sin(i*dx);
    y[i] = 0.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, dx, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;

  for (int i = 0; i < N-1; i++) {
    maxError = fmax(maxError, fabs(y[i]-cos((i + 0.5f)*dx)));
//    std::cout << i*dx << " " << y[i] << " " << maxError << std::endl;
  }


  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
