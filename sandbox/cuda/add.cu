
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int a0, int a1, int s0, int s1, int d0, int d1, float dx, float *x, float *y)
{ 
  int delta = a0*a1 / (blockDim.x * gridDim.x) + 1;
  int start = delta*(blockIdx.x * blockDim.x + threadIdx.x);
  int end = start + delta;
  if (end >= a0*a1) {
    end =  a0*a1 - 1;
  }
  
  int p0 = start % a0 + s0;
  int q0 = start / a0;
  int p1 = q0 % a1 + s1;

  int skip0 = d0 - s0 - a0;
  int t0 = s0 + a0;

  int j = p0 + d0*p1;

  int i = start;
  while (i < end) {
    
    y[j] = (x[j+1] + x[j-1] + x[j+d0] + x[j-d0] - 4*x[j]) / (dx*dx);
    
    ++i;
    if (++j >= t0) {
      j += skip0 - 1;
    }
  }
}

int main(void)
{
  int D = 1<<10;
  int N = D*D;
  float *x, *y;
  float dx = float(4.0f/D);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < D; ++i) {
    for (int j = 0; j < D; ++j) {
      float r2 = (i*i + j*j)*dx*dx;
      x[i*D + j] = exp(-r2);
    }
  }

  for (int i = 0; i < N; ++i) {
    y[i] = 0.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(D - 8, D - 8, 4, 4, D, D, dx, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = -1.0f;
  int maxI, maxJ;

  for (int i = 0; i < D; ++i) {
    for (int j = 0; j < D; ++j) {
      float r2 = (i*i + j*j)*dx*dx;
      float expected = (i<4 || j<4 || i>=D-4 || j>=D-4) ? 0.0f : 4.0f*(r2 - 1)*exp(-r2);
      
      float err = fabs(y[i*D + j] - expected);
      if (err > maxError) {
        maxError = err;
        maxI = i;
        maxJ = j;
      }
//      std::cout << i*dx << " " << j*dx << " " << y[i*D + j] << std::endl;
    }
  }

  std::cout << "Max error: " << maxError << " " << maxI << " " << maxJ << " " << y[maxI*D + maxJ] << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
