
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

template<int rank>
struct Array {
  int val[rank];
};

namespace device {
  template<typename T>
  __device__
  const T& min(const T& a, const T& b) {
    return (a>b) ? b : a;
  }
}

template<int rank>
struct GridStride {
  int innerDim[rank];
  int offset[rank];
  int outerDim[rank];

  __host__ __device__
  int getInnerCount() const {
    int result = innerDim[0];
    for (int i=1; i<rank; ++i) {
      result *= innerDim[i];
    }
    return result;
  }

  __host__ __device__
  void innerPosFromInnerCount(int count, int (&pos)[rank]) const {
    for (int i=rank-1; i>=0; --i) {
      pos[i] = count % innerDim[i];
      count = count / innerDim[i];
    }
  }

  __host__ __device__
  int outerCountFromInnerPos(int pos[rank]) const {
    int count = pos[0] + offset[0];
    for (int i=1; i<rank; ++i) {
      count = count*outerDim[i-1] + pos[i] + offset[i];
    } 
    return count;
  }
};

template<int rank>
struct LocalGridIterator {
  int nIter;
  int delta;
  int start;
  int end;
  int i;
  int j;
  int skip0;
  int t0;

  __device__
  LocalGridIterator(const GridStride<rank> &stride) {
    nIter = stride.getInnerCount();
    delta = nIter / (blockDim.x * gridDim.x) + 1;
    start = delta*(blockIdx.x * blockDim.x + threadIdx.x);
    end = device::min(start + delta, nIter);

    int innerPos[2];
    stride.innerPosFromInnerCount(start, innerPos);

    skip0 = stride.outerDim[0] - stride.innerDim[0];
    t0 = stride.offset[0] + stride.innerDim[0]
      + stride.outerDim[0]*(stride.offset[1] + innerPos[1]);


    j = stride.outerCountFromInnerPos(innerPos);
    i = start;
  }
};

// Kernel function to add the elements of two arrays
__global__
void add(GridStride<2> stride, float dx, float *x, float *y)
{ 
  LocalGridIterator<2> iter(stride);

  while (iter.i < iter.end) {
    
    y[iter.j] = (x[iter.j+1] + x[iter.j-1] + x[iter.j+stride.outerDim[0]] + x[iter.j-stride.outerDim[0]] - 4*x[iter.j]) / (dx*dx); 
    
    ++iter.i;
    if (++iter.j >= iter.t0) {
      iter.j += iter.skip0;
      iter.t0 += stride.outerDim[0];
    }
  }
}

template<typename T>
T testFunc(T x, T y) {
  float r2 = x*x + y*y;
  return exp(-r2);
//    return 10000*x + y;
}

int main(void)
{
  int D = 200;
  int N = D*D;
  float *x, *y;
  float dx = float(0.5f/D);

  GridStride<2> stride{{D - 8, D - 8}, {4, 4}, {D, D}};

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < D; ++i) {
    for (int j = 0; j < D; ++j) {
      x[i*D + j] = testFunc(i*dx, j*dx);
    }
  }

  for (int i = 0; i < N; ++i) {
    y[i] = 0.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<2, 1>>>(stride, dx, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors
  float maxError = -1.0f;
  float comparedTo;
  int maxI, maxJ;

  for (int i = 0; i < D; ++i) {
    for (int j = 0; j < D; ++j) {
//      float r2 = (i*i + j*j)*dx*dx;
//      float expected = (i<4 || j<4 || i>=D-4 || j>=D-4) ? 0.0f : 4.0f*(r2 - 1)*exp(-r2);
      float expected = (i<4 || j<4 || i>=D-4 || j>=D-4) ? 0.0f : (
        testFunc<float>(i*dx - dx, j*dx) + testFunc<float>(i*dx + dx, j*dx) + 
        testFunc<float>(i*dx, j*dx - dx) + testFunc<float>(i*dx, j*dx + dx) 
        - 4.0*testFunc(i*dx, j*dx)
      ) /(dx*dx);
    
      float err = fabs(y[i*D + j] - expected);
      if (err > maxError) {
        maxError = err;
        comparedTo = expected;
        maxI = i;
        maxJ = j;
      }
      std::cout << i*dx << " " << j*dx << " " << y[i*D + j] << std::endl;
    }
    std::cout << std::endl;
  }

  std::cerr << "Max error: " << maxError << " " << maxI << " " << maxJ << " " << y[maxI*D + maxJ] << " " << comparedTo << std::endl;

//   float xf = maxI*dx;
//   float yf = maxJ*dx;

//   float expectedAtMax = // testFunc(xf - dx, yf); 
// ( testFunc(xf - dx, yf) + testFunc(xf + dx, yf) + 
//   testFunc(xf, yf - dx) + testFunc(xf, yf + dx)
//   - 4.0*testFunc(xf, yf)
// )/(dx*dx);

//   std::cout << "Expected value: " << expectedAtMax << std::endl;

//   // double check, get it? ;)
//   double xd = maxI*dx;
//   double yd = maxJ*dx;
  
//   double precise = // testFunc(xd - dx, yd); 
// ( testFunc(xd - dx, yd) + testFunc(xd + dx, yd) + 
//   testFunc(xd, yd - dx) + testFunc(xd, yd + dx) 
//   - 4.0*testFunc(xd, yd)
// )/(double(dx)*double(dx));

//   std::cout << "Better value: " << precise << std::endl;
  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
